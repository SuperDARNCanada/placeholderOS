
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h> //cuComplex type and all cuCmulf/cuCaddf functions.
#include <iostream>
#include <stdint.h>

/*Overloaded __shfl_down function. Default does not recognize cuComplex but
does for equivalent float2 type.
https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
http://docs.nvidia.com/cuda/cuda-c-programming-guide/#warp-shuffle-functions
*/
__device__ inline hipComplex __shfl_down(hipComplex var, unsigned int srcLane, int width=32){
    float2 a = *reinterpret_cast<float2*>(&var);
    a.x = __shfl_down(a.x, srcLane, width); // REVIEW #0 Does this call the original function since the a variable is a float now?
                                            // REPLY Overloading works by allowing the compiler to detect which version of the function to use for different types, so yes.
                                            // This function just shuffles down each real and imag part seperately like how it was done in the linked example.
    a.y = __shfl_down(a.y, srcLane, width);
    return *reinterpret_cast<hipComplex*>(&a);
}

/*Slightly modified version of reduction #5 from NVIDIA examples
/usr/local/cuda/samples/6_Advanced/reduction
*/
__device__ hipComplex parallel_reduce(hipComplex* data, uint32_t tap_offset) { // REVIEW #28 can tap_offset ever be negative? Maybe should make it uint32_t
                                                                        // REPLY tap_offset is always based off threadIdx which is never negative. I will change this to uint32_t though for consistency

    auto filter_tap_num = threadIdx.x;
    auto num_filter_taps = blockDim.x;
    hipComplex total_sum = data[tap_offset];


    if ((num_filter_taps >= 512) && (filter_tap_num < 256))
    {
        total_sum = hipCaddf(total_sum,data[tap_offset  + 256]);
        data[tap_offset] = total_sum; // REVIEW #25 Is it necessary for speed to have two '=' statements on one line? it took a while to see the second one, therefore more confusing. split into two lines
    }                                       // REPLY okay

    __syncthreads();

    if ((num_filter_taps >= 256) && (filter_tap_num < 128))
    {
        total_sum = hipCaddf(total_sum, data[tap_offset + 128]);
        data[tap_offset] = total_sum;
    }

     __syncthreads();

    if ((num_filter_taps >= 128) && (filter_tap_num <  64))
    {
        total_sum = hipCaddf(total_sum, data[tap_offset  +  64]);
        data[tap_offset] = total_sum;
    }

    __syncthreads();

    if ( filter_tap_num < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (num_filter_taps >=  64) total_sum = hipCaddf(total_sum, data[tap_offset + 32]);
        // Reduce final warp using shuffle // REVEW #3 This code depends upon a warp all executing threads at exactly the same time, if it didn't then double the total_sum value for the second half of the threads would be accidentally used. Can be explicit by putting if statement in the for loop [if (filter_tap_num < offset) we think]
                                           // REPLY The CUDA examples dont use that. I think shfl is designed to mark threads inactive if the source lane as they say wraps out of bounds.
                                            //It also happens simultaneously eliminating the need for __syncthreads
        for (int offset = warpSize/2; offset > 0; offset /= 2) // REVIEW #0 Where does warpSize come from? Don't you need to get it from the gpu_properties?
                                                               // REPLY http://docs.nvidia.com/cuda/cuda-c-programming-guide/#built-in-variables
        {
            total_sum = hipCaddf(total_sum,__shfl_down(total_sum, offset)); // REVIEW #3 Very not-obvious. Seems like it needs to know that total_sum is the variable/memory to work on, need a comment to tell us how this works
        }                                                                  // REPLY This __shfl construct was designed non obvious. But it just consistently halves the threads and shuffles a register value to the active threads.
    }

    return total_sum;
}

__global__ void decimate1024(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel) //REVIEW #1 describe thread/block/grid dimensions and indices
{                                 //REPLY docstring

    extern __shared__ hipComplex filter_products[]; // REVIEW #4 comment why is this extern and why is it necessary to be dynamically allocated?
                                                   //REPLY Shared memory needs to be dynamically allocated http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#shared

    auto channel_num = blockIdx.y;
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.x + threadIdx.x; // REVIEW #0 should be blockDim.x
                                                              // REPLY correct

    //If an offset should extend past the length of samples per antenna
    //then zeroes are used as to not segfault or run into the next buffer.
    //output samples using these will be discarded as to not introduce edge effects
    hipComplex sample;
    if ((dec_sample_offset + threadIdx.x) >= samples_per_channel) {
        sample = make_hipComplex(0.0f,0.0f); // REVIEW #1 explain zero-padding, #0, correct this after to throw out edge effects (per stage) ceil((num_samps - num_taps)/dm_rate)
    }                                       // REPLY okay. We decided samples will be corrected at the end so this can stay for now
    else {
        auto final_offset = channel_offset + dec_sample_offset + threadIdx.x;
        sample = original_samples[final_offset];
    }


    filter_products[tap_offset] = hipCmulf(sample,filter_taps[tap_offset]); // REVIEW #4 tell user that this comes from cuComplex.h, any side effects?
                                                                           // REPLY dont think so.
    __syncthreads(); // REVIEW #1 Synchronizes all threads in a block, meaning 1 output sample per rx freq is ready to be calculated with the parallel reduce

    auto calculated_output_sample = parallel_reduce(filter_products, tap_offset); // REVIEW #26 Should this be called something like 'decimated_sample' instead - to indicate that it is going into the array of decimated samples? total_sum could be the variable name in parallel reduce, but not here in this context
                                                                   // REPLY alright

    //this is all wrong now that I think about it. At least for second and third stage
    if (threadIdx.x == 0) { // REVIEW #1 Explain how you're setting up the array of decimated samples
        channel_offset = channel_num * samples_per_channel/dm_rate; // REVIEW #13 gridDimx is already samples_per_channel/dm_rate, use it instead
                                                                    // REPLY it is, yet I've been trying to avoid direct use of CUDA builtin types since it not obvious what they mean
        auto total_channels = blockDim.y; // REVIEW #0 This should be gridDim.y if you intend to use 'total_channels' as antennas (should use 'antennas')
        auto freq_offset = threadIdx.y * total_channels; // REVIEW #0 still need to multiply by gridDim.x here to get index into proper location
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = calculated_output_sample;
    }
}

__global__ void decimate2048(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel)
{

    extern __shared__ hipComplex filter_products[];

    auto channel_num = blockIdx.y; // REVIEW #26 -Again here channels/freqs/antennas is confused and needs to be consistent, maybe we avoid the word 'channel' altogether
    auto channel_offset = channel_num * samples_per_channel;

    auto dec_sample_num = blockIdx.x;
    auto dec_sample_offset = dec_sample_num * dm_rate;

    auto tap_offset = threadIdx.y * blockDim.y + 2 * threadIdx.x; //REVIEW #0 should be blockDim.x

    hipComplex sample_1;
    hipComplex sample_2;
    if ((dec_sample_offset + 2 * threadIdx.x) >= samples_per_channel) {
        sample_1 = make_hipComplex(0.0,0.0);
        sample_2 = make_hipComplex(0.0,0.0);
    }
    else {
        auto final_offset = channel_offset + dec_sample_offset + 2*threadIdx.x;
        sample_1 = original_samples[final_offset];
        sample_2 = original_samples[final_offset+1];  // REVIEW #0 what if final_offset = samples_per_channel - 1 so that sample_1 is in bounds but sample_2 is out of bounds
    }


    filter_products[tap_offset] = hipCmulf(sample_1,filter_taps[tap_offset]); //
    filter_products[tap_offset+1] = hipCmulf(sample_2, filter_taps[tap_offset+1]); // REVIEW #0 what if you have an odd number of taps so that in the last thread filter_taps[tap_offset+1] isn't defined ? (unless all filters are of length 2^x)
                                                                                  // REPLY filters will have to be power of 2 in length for this to work.
    filter_products[tap_offset] = hipCaddf(filter_products[tap_offset],filter_products[tap_offset+1]);

    __syncthreads();
    auto calculated_output_sample = parallel_reduce(filter_products, tap_offset); // REVIEW #0 pass new variable for offset in filter products so you are not passing only even values
                                                                  // REPLY this can stay the same, but I missed a line of code that needs to happen first

    //again, wrong for now.
    if (threadIdx.x == 0) { // REVIEW #1 Explain how you're setting up the array of decimated samples
        channel_offset = channel_num * samples_per_channel/dm_rate; // REVIEW #13 gridDimx is already samples_per_channel/dm_rate, use it instead
        auto total_channels = blockDim.y; // REVIEW #0 This should be gridDim.y if you intend to use 'total_channels' as antennas (should use 'antennas')
        auto freq_offset = threadIdx.y * total_channels; // REVIEW #0 still need to multiply by gridDim.x here to get index into proper location
        auto total_offset = freq_offset + channel_offset + dec_sample_num;
        decimated_samples[total_offset] = calculated_output_sample;
    }
}

static dim3 create_grid(uint32_t num_samples, uint32_t dm_rate, uint32_t num_channels) // REVIEW #26 no more channels
{
    auto num_blocks_x = num_samples/dm_rate;
    auto num_blocks_y = num_channels;
    auto num_blocks_z = 1;
    std::cout << "    Grid size: " << num_blocks_x << " x " << num_blocks_y << " x "
        << num_blocks_z << std::endl;
    dim3 dimGrid(num_blocks_x,num_blocks_y,num_blocks_z);

    return dimGrid;
}

static dim3 create_block(uint32_t num_taps, uint32_t num_freqs)
{
    auto num_threads_x = num_taps;
    auto num_threads_y = num_freqs;
    auto num_threads_z = 1;
    std::cout << "    Block size: " << num_threads_x << " x " << num_threads_y << " x "
        << num_threads_z << std::endl;
    dim3 dimBlock(num_threads_x,num_threads_y,num_threads_z);

    return dimBlock;
}

void decimate1024_wrapper(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, hipStream_t stream) { // REVIEW #1 describe how this works including choice of blocks and grids

    auto shr_mem_taps = num_freqs * num_taps * sizeof(hipComplex); // REVIEW #32 why do we need this?
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels);
    auto dimBlock = create_block(num_taps,num_freqs);
    decimate1024<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
                filter_taps, dm_rate, samples_per_channel);

}

void decimate2048_wrapper(hipComplex* original_samples,
    hipComplex* decimated_samples,
    hipComplex* filter_taps, uint32_t dm_rate,
    uint32_t samples_per_channel, uint32_t num_taps, uint32_t num_freqs,
    uint32_t num_channels, hipStream_t stream) {

    auto shr_mem_taps = num_freqs * num_taps * sizeof(hipComplex);
    std::cout << "    Number of shared memory bytes: "<< shr_mem_taps << std::endl;

    auto dimGrid = create_grid(samples_per_channel, dm_rate, num_channels);
    auto dimBlock = create_block(num_taps/2, num_freqs);
    decimate2048<<<dimGrid,dimBlock,shr_mem_taps,stream>>>(original_samples, decimated_samples,
        filter_taps, dm_rate, samples_per_channel);
}